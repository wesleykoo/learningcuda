#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#define M 256
#define K 512
#define N 256
#define BLOCK_SIZE 32

// CPU matrix multiplication
void matmul_cpu(float *a, float *b, float *c, int m, int k, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            c[i * n + j] = 0;
            for (int l = 0; l < k; l++) {
                c[i * n + j] += a[i * k + l] * b[l * n + j];
            }
        }
    }
};

// CUDA kernel for matrix multiplication
__global__ void matmul_gpu(float *a, float *b, float *c, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0;
        for (int i = 0; i < k; i++) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
};

// initialize matrix
void init_matrix(float *matrix, int row, int col) {
    for (int i = 0; i < size; i++) {
        matrix[i] = (float)rand() / RAND_MAX;
    }
}

// function to measure time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

