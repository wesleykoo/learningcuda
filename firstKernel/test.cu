
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void whoami(void) {
    int block_id = blockIdx.x + 
    blockIdx.y * gridDim.x + 
    blockIdx.z * gridDim.x * gridDim.y;

    int block_offset = block_id * 
    blockDim.x * 
    blockDim.y * 
    blockDim.z;

    int thread_offset = 
    threadIdx.x + 
    threadIdx.y * blockDim.x + 
    threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;
    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id, 
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);

}

int main(int argc, char **argv) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;
    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("blocks_per_grid: %d\n", blocks_per_grid);
    printf("threads_per_block: %d\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z);
    dim3 threadsPerBlock(t_x, t_y, t_z);

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}